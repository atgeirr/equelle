#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>

//#include <string>
//#include <fstream>
//#include <iterator>
#include <hip/hip_runtime.h>

#include <stdlib.h>

//#include "EquelleRuntimeCUDA.hpp"
#include "EquelleRuntimeCUDA_cuda.hpp"

// Implementation of the class CollOfScalar

CollOfScalar::CollOfScalar()
{
    // Blank
    values = 0;
    size = 0;
    dev_values = 0;
    //dev_vec = thrust::device_vector<double>(0);

}

CollOfScalar::CollOfScalar(int size) {
    // dev_vec.reserve(size);
    this->size = size;
    values = (double*)malloc(size*sizeof(double));
    //dev_vec = thrust::device_vector<double>(size);
    hipError_t status = hipMalloc( (void**)&dev_values, size*sizeof(double));
    if ( status != hipSuccess ) {
	std::cout << "Error allocating dev_values in CollOfScalar(int)\n";
	exit(0);
    }
}


// Copy constructor
CollOfScalar::CollOfScalar(const CollOfScalar& coll) {
    std::cout << "Copy constructor!\n";
    size = coll.size;
    values = 0;
    dev_values = 0;
    if (coll.values != 0) {
	values = (double*)malloc(size*sizeof(double));
	for ( int i = 0; i < size; i++) {
	    values[i] = coll.values[i];
	}
    }
    if (coll.dev_values != 0) {
	hipError_t status = hipMalloc( (void**)&dev_values, size*sizeof(double));
	if ( status != hipSuccess ) {
	    std::cout << "Error allocating dev_values in CollOfScalar(CollOfScalar)\n";
	    exit(0);
	}
	status = hipMemcpy(dev_values, coll.dev_values, size*sizeof(double),
			    hipMemcpyDeviceToDevice);
	if ( status != hipSuccess ){
	    std::cout << "Error copying dev_values in copy constructor\n";
	    exit(0);
	}
    }    
}


// Destructor:
CollOfScalar::~CollOfScalar() {
    if ( size > 0 ) {
	size = 0;
    }
    if (values != 0) {
	std::cout << "Freeing values\n";
	free(values);
	//values = 0;
    }
    if (dev_values != 0) {
	hipError_t status = hipFree(dev_values);
	if (status != hipSuccess) {
	    std::cout << "Error cuda-freeing in destructor of CollOfScalar\n";
	    std::cout << "\tError code: " << hipGetErrorString(status) << std::endl;
	    exit(0);
	}
	//dev_values = 0;
    }
}

double* CollOfScalar::getDevValues() const {
    return dev_values;
}

//double* CollOfScalar::getHostValues() const {
//    return values;
//}

void CollOfScalar::copyToHost() const
{
    std::cout << "copyToHost() - dev_values = " << dev_values << std::endl;

    hipError_t hipError_t = hipMemcpy( values, dev_values, size*sizeof(double),
					hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess) {
	std::cout << "Error copying to host in output. \n\tError code = ";
	std::cout << hipGetErrorString(hipError_t) << "\n";
	exit(0);
    }

}

double CollOfScalar::getValue(int index) const
{
    if ( index > -1 && index < size) {
	return values[index];
    }
    else {
	exit(0);
	return 0;
    }
}

void CollOfScalar::setValue(int index, double value)
{
    if (index > -1 && index < size) {
	values[index] = value;
    }
}

void CollOfScalar::setValuesFromFile(std::istream_iterator<double> begin,
				     std::istream_iterator<double> end)
{
    thrust::host_vector<double> host_vec(begin, end);
    //for( std::istream_iterator<double> i = begin; i != end; i++) {
    //dev_vec.insert(dev_vec.begin(), begin, end);
    //}
    //double* a;
    //hipError_t t = hipMalloc( (void**)&a, sizeof(double)*dev_vec.size());
    for(int i = 0; i < host_vec.size(); i++) {
	values[i] = host_vec[i];
    }
    //dev_vec = host_vec;
    hipError_t cudaStatus = hipMemcpy( dev_values, values, size*sizeof(double),
					 hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
	std::cout << "Error in hipMemcpy to dev from file.\n";
	exit(0);
    }
}

void CollOfScalar::setValuesUniform(double val, int size)
{
    //dev_vec.reserve(size);
    //thrust::fill(dev_vec.begin(), dev_vec.begin() + 4, val);
    // dev_vec.insert(dev_vec.begin(), dev_vec.begin() + size, val);
       //dev_vec.push_back(1.0);
}

int CollOfScalar::getSize() const
{
    //return dev_vec.size();
    return size;
}
