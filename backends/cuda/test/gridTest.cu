

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>

#include <iostream>
#include <exception>


#include "deviceGrid.hpp"

#include "gridTest.h"


using namespace equelleCUDA;

// Test functions return 0 for success, 1 otherwise

// This test expect 4x3 grid

int all_cells_test(DeviceGrid dg) {
    //Expect the Collection to be full.

    Collection coll = dg.allCells();
    if ( !coll.isFull() ) {
	std::cout << "Error in gridTest.cu - all_cells_test\n";
	return 1;
    }
    std::cout << "Passed all_cells_test\n";
    return 0;

}

int all_faces_test(DeviceGrid dg) {
    // Expect the Collection to be full
    Collection coll = dg.allFaces();
    if ( !coll.isFull() ) {
	std::cout << " Error in gridTest.cu - all_faces_test\n";
	return 1;
    }
    std::cout << "Passed all_faces_test\n";
    return 0;
}

int boundary_faces_test(DeviceGrid dg) {
    // Expecting a non-full collection containing
    // { 0 4 5 9 10 14 15 16 17 18 27 28 29 30 }
    int lf[] = {0, 4, 5, 9, 10, 14, 15, 16, 17, 18, 27, 28, 29, 30 };
    int lf_size = 14;
    Collection coll = dg.boundaryFaces();
    if ( coll.isFull() ) {
	std::cout << "Error in gridTest.cu - boundary_faces_test\n";
	std::cout << "\tCollection should not be full\n";
	return 1;
    }

    thrust::host_vector<int> host = coll.toHost();
    std::cout << "Boundary faces is the following:\n";
    bool correct = true;
    for (int i = 0; i < host.size(); ++i) {
	std::cout << host[i] << " ";
	if (i < lf_size) {
	    if (host[i] != lf[i]) {
		correct = false;
	    }
	}
    }
    if (correct) {
	std::cout << "\n\tThis is correct\n";
    } else {
	std::cout << "\n\tThis is wrong\n";
	std::cout << "Error in gridTest.cu - boundary_faces_test\n";
	std::cout << "\tThe indices in the collection is wrong\n";
	return 1;
    }

    std::cout << "Testing size\n";
    if ( coll.size() != lf_size ) {
	std::cout << "Error in gridTest.cu - boundary_faces_test\n";
	std::cout << "\tThe collection is of wrong size!\n";
    }
    
    return 0;
}

int cuda_main(DeviceGrid dg) {
    
    std::cout << "From cuda_main!\n";

    DeviceGrid dg2(dg);


    std::cout << "Test:  (4?) " << dg.setID(1)  << std::endl;
    std::cout << "Test2: (4?) " << dg2.setID(2) << std::endl;

    if ( all_cells_test(dg) ) {
	return 1;
    }
    if ( all_faces_test(dg) ) {
	return 1;
    }
    if ( boundary_faces_test(dg) ) {
	return 1;
    }

    return 0;
}



// more to come!
