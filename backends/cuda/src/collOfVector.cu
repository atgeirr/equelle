#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <opm/core/utility/ErrorMacros.hpp>
#include <iostream>
#include <vector>


#include "CollOfVector.hpp"
#include "CollOfScalar.hpp"


using namespace equelleCUDA;


CollOfVector::CollOfVector() 
    : CollOfScalar()
{
    // intentionally left blank
}

CollOfVector::CollOfVector(const int size, const int dim)
    : CollOfScalar(size*dim), dim_(dim)
{
    // intentionally left blank
}

CollOfVector::CollOfVector(const std::vector<double>& host, const int dim)
    : CollOfScalar(host), dim_(dim)
{
    // intentionally left blank
}


// Copy-constructor
CollOfVector::CollOfVector(const CollOfVector& coll)
    : CollOfScalar(coll), dim_(coll.dim_)
{
    // intentionally left blank
}
  

//Operator []
CollOfScalar CollOfVector::operator[](const int index) const {
    
    if ( index < 0 || index >= dim_) {
	OPM_THROW(std::runtime_error, "Illigal dimension index " << index << " for a vector of dimension " << dim_);
    }
    
    CollOfScalar out(size()/dim_);
    
    dim3 block(out.block());
    dim3 grid(out.grid());

    collOfVectorOperatorIndexKernel<<<grid,block>>>( out.data(),
						     this->data(),
						     out.size(),
						     index,
						     dim_);
						    
    return out;
}

  

__global__ void equelleCUDA::collOfVectorOperatorIndexKernel( double* out,
							      const double* vec,
							      const int size_out,
							      const int index,
							      const int dim)
{
    // Index:
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if ( i < size_out ) {
	out[i] = vec[i*dim + index];
    }
}