#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/detail/raw_pointer_cast.h>

#include "wrapEquelleRuntime.hpp"
#include "CollOfScalar.hpp"
#include "equelleTypedefs.hpp"

using namespace equelleCUDA;

// Have already performed a check on sizes.
CollOfScalar equelleCUDA::trinaryIfWrapper( const CollOfBool& predicate,
					    const CollOfScalar& iftrue,
					    const CollOfScalar& iffalse) {
    CollOfScalar out(iftrue.size());
    const bool* pred_ptr = thrust::raw_pointer_cast( &predicate[0] );
    dim3 block(iftrue.block());
    dim3 grid(iftrue.grid());
    trinaryIfKernel<<<grid,block>>>(out.data(),
				    pred_ptr,
				    iftrue.data(),
				    iffalse.data(),
				    iftrue.size());
    return out;
    //return CollOfScalar(predicate.size(), 0);
}


__global__ void equelleCUDA::trinaryIfKernel( double* out,
					      const bool* predicate,
					      const double* iftrue,
					      const double* iffalse,
					      const int size) 
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ( index < size) {
	double temp;
	if (predicate[index]) {
	    temp = iftrue[index];
	}
	else {
	    temp = iffalse[index];
	}
	out[index] = temp;
    }
}