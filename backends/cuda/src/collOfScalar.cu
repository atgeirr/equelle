#include "hip/hip_runtime.h"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/fill.h>
//#include <thrust/copy.h>
//#include <thrust/sequence.h>

#include <string>
//#include <fstream>
//#include <iterator>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <vector>
#include <iostream>

// For error exception macro:
#include <opm/core/utility/ErrorMacros.hpp>

#include "CollOfScalar.hpp"
#include "DeviceGrid.hpp"
#include "CollOfIndices.hpp"


// Implementation of the class CollOfScalar

using namespace equelleCUDA;


CollOfScalar::CollOfScalar() 
    : size_(0), 
      dev_values_(0),
      block_x_(0),
      grid_x_(0),
      setup_(0)
#ifdef EQUELLE_DEBUG
    , debug_vec_(0)
#endif // EQUELLE_DEBUG
{
    // Intentionally left blank
}

// Allocating memory without initialization
CollOfScalar::CollOfScalar(const int size) 
    : size_(size),
      dev_values_(0),
      block_x_(equelleCUDA::MAX_THREADS),
      grid_x_((size_ + block_x_ - 1) / block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(size,0)
#endif // EQUELLE_DEBUG
{
    cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
    checkError_("hipMalloc in CollOfScalar::CollOfScalar(int)");
#ifdef EQUELLE_DEBUG
    std::cout << "Debug mode is on!\n";
#endif // EQUELLE_DEBUG
}

CollOfScalar::CollOfScalar(const int size, const double value) 
    : size_(size),
      dev_values_(0),
      block_x_(equelleCUDA::MAX_THREADS),
      grid_x_((size_ + block_x_ - 1) / block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(size, value)
#endif // EQUELLE_DEBUG
{
    // Can not use hipMemset as it sets float values on a given
    // number of bytes.
    std::cerr << __PRETTY_FUNCTION__ << std::endl;

    std::vector<double> host_vec(size_, value);

    cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
    checkError_("hipMalloc in CollOfScalar::CollOfScalar(int, double)");
        
    cudaStatus_ = hipMemcpy(dev_values_, &host_vec[0], size_*sizeof(double),
				    hipMemcpyHostToDevice);
    checkError_("hipMemcpy in CollOfScalar::CollOfScalar(int, double)");

} 


// Constructor from vector, in order to do testing
CollOfScalar::CollOfScalar(const std::vector<double>& host_vec)
    : size_(host_vec.size()),
      dev_values_(0),
      block_x_(equelleCUDA::MAX_THREADS),
      grid_x_((size_ + block_x_ - 1) / block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(host_vec)
#endif // EQUELLE_DEBUG
{
    cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
    checkError_("hipMalloc in CollOfScalar::CollOfScalar(std::vector<double>)");
    
    cudaStatus_ = hipMemcpy(dev_values_, &host_vec[0], size_*sizeof(double),
			    hipMemcpyHostToDevice);
    checkError_("hipMemcpy in CollOfScalar::CollOfScalar(std::vector<double>)");
}


// Copy constructor
CollOfScalar::CollOfScalar(const CollOfScalar& coll) 
    : size_(coll.size_), 
      dev_values_(0),
      grid_x_(coll.grid_x_),
      block_x_(coll.block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(coll.size_, 0)
#endif // EQUELLE_DEBUG
{
    std::cout << "Copy constructor!\n";
    std::cerr << __PRETTY_FUNCTION__ << std::endl;    

    if (coll.dev_values_ != 0) {
	cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
	checkError_("hipMalloc in CollOfScalar::CollOfScalar(const CollOfScalar&)"); 

	cudaStatus_ = hipMemcpy(dev_values_, coll.dev_values_, size_*sizeof(double),
				 hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy in CollOfScalar::CollOfScalar(const CollOfScalar&)");
    }
    
#ifdef EQUELLE_DEBUG
    // Copy value to the std::vector debug_vec_
    std::cout << "\tDEBUG IS ON!\n";
    if (coll.dev_values_ != 0 ) {
	cudaStatus_ = hipMemcpy( &debug_vec_[0], dev_values_, size_*sizeof(double),
				  hipMemcpyDeviceToHost );
	checkError_("hipMemcpy for DEBUG in CollOfScalar::CollOfScalar(const CollOfScalar&)");
	last_val = debug_vec_[size_ - 1];
    }
#endif // EQUELLE_DEBUG
}


// Copy assignment operator
CollOfScalar& CollOfScalar::operator= (const CollOfScalar& other) {
    std::cerr << __PRETTY_FUNCTION__ << std::endl;    

    // Protect agains " var = var " , self assignment
    if ( this != &other ) {

	// First idea: Make this->dev_values_ point to other.dev_values_
	// and set other.dev_values_ = 0.
	// Why is this a bad idea? We should still be able to use other.
	// THEREFORE: Need to copy the content of other.dev_values_ to
	// this->dev_values_.

	// this->dev_values will be overwritten, and can safely be freed,
	// But if the collections are of the same size (likely) we
	// Will just overwrite the old values.

	std::cout << "COPY ASSIGNMENT OPERATOR ( this->size = " << this->size_ << ")\n";

	if ( this->size_ != other.size_) {

	    std::cout << "\tCHANGING SIZE FROM " << this->size_ << " TO " << other.size_ << "\n";
		
	    // If different size: Is this even allowed?
	    // Free memory:
	    cudaStatus_ = hipFree(this->dev_values_);
	    checkError_("hipFree(this->dev_values_) in CollOfScalar::operator=(const CollOfScalar&)");
	    // Allocate new memory:
	    cudaStatus_ = hipMalloc((void**)&this->dev_values_,
				     sizeof(double) * other.size_);
	    checkError_("hipMalloc(this->dev_values_) in CollOfScalar::operator=(const CollOfScalar&)");
	    
	    // Set variables depending on size_:
	    this->size_ = other.size_;
	    this->block_x_ = other.block_x_;
	    this->grid_x_ = other.grid_x_;
	}

	// Copy memory block from other to this:
	cudaStatus_ = hipMemcpy( this->dev_values_, other.dev_values_,
				  sizeof(double) * this->size_,
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(dev_values_) in CollOfScalar::operator=(const CollOfScalar&)");
	
#ifdef EQUELLE_DEBUG
	if ( debug_vec_.size() != this->size_) {
	    std::cout << "\t\tDebug vector is of size " << debug_vec_.size() << 
		" while this->size_ is " << this->size_ << "\n";
	    std::vector<double> temp(this->size_, 0);
	    cudaStatus_ = hipMemcpy( &temp[0], other.dev_values_,
				      sizeof(double) * this->size_,
				      hipMemcpyDeviceToHost);
	    checkError_("hipMemcpy(temp) in CollOfScalar::operator=(const CollOfScalar&)");
	    debug_vec_ = temp;
	}
	else {
	    cudaStatus_ = hipMemcpy( &debug_vec_[0], other.dev_values_,
				      sizeof(double) * this->size_,
				      hipMemcpyDeviceToHost);
	    checkError_("hipMemcpy(debug_vec) in CollOfScalar::operator=(const CollOfScalar&)");
	}
	last_val = debug_vec_[size_-1];
#endif // EQUELLE_DEBUG


    } // if this != &other
    
    return *this;

} // Assignment copy operator!



// Destructor:
CollOfScalar::~CollOfScalar() {
    if (dev_values_ != 0) {
	cudaStatus_ = hipFree(dev_values_);
	checkError_("hipFree in CollOfScalar::~CollOfScalar");
    }
}

#ifdef EQUELLE_DEBUG
// Debug function to get all values to host so that they can be seen by e.g. qtcreator
void CollOfScalar::debug() const {
    cudaStatus_ = hipMemcpy( &debug_vec_[0], dev_values_, sizeof(double)*size_,
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(debug_vec_) in CollOfScalar::debug()");
    last_val = debug_vec_[size_ - 1];
}
#endif // EQUELLE_DEBUG



const double* CollOfScalar::data() const {
    return dev_values_;
}

double* CollOfScalar::data() {
    return dev_values_;
}




kernelSetup CollOfScalar::setup() const {
    return setup_;
}

// Assumes that values are already allocated on host
std::vector<double> CollOfScalar::copyToHost() const
{
    //std::cout << "copyToHost() - val_ptr = " << dev_values << std::endl;
    
    // Fill host_vec with zeros:
    std::vector<double> host_vec(size_, 0);

    cudaStatus_ = hipMemcpy( &host_vec[0], dev_values_, size_*sizeof(double),
			     hipMemcpyDeviceToHost);
    checkError_("hipMemcpy in CollOfScalar::copyToHost");
    
    return host_vec;
}


int CollOfScalar::size() const
{
    return size_;
}



void CollOfScalar::checkError_(const std::string& msg) const {
    if ( cudaStatus_ != hipSuccess ) {
	OPM_THROW(std::runtime_error, "\nCuda error\n\t" << msg << " - Error code: " << hipGetErrorString(cudaStatus_));
	//std::cout <<  "Cuda error\n\t" << msg << "\n\tError code: " << hipGetErrorString(cudaStatus) << std::endl;
	//exit(0);
    }
}






/////////////////////////////////////////////////////////////////////////////////
/// ----------------------- OPERATOR OVERLOADING: -----------------------------//
/////////////////////////////////////////////////////////////////////////////////



CollOfScalar equelleCUDA::operator-(const CollOfScalar& lhs, const CollOfScalar& rhs) {

    CollOfScalar out = lhs;
    //double* lhs_dev = lhs.data();
    const double* rhs_dev = rhs.data();
    double* out_dev = out.data();

    std::cout << "Calling minus_kernel!\n";
    kernelSetup s = out.setup();
    minus_kernel <<<s.grid, s.block>>>(out_dev, rhs_dev, out.size());
    return out;
}

CollOfScalar equelleCUDA::operator+(const CollOfScalar& lhs, const CollOfScalar& rhs) {

    CollOfScalar out = lhs;
    const double* rhs_dev = rhs.data();
    double* out_dev = out.data();

    kernelSetup s = out.setup();
    plus_kernel <<<s.grid, s.block>>>(out_dev, rhs_dev, out.size());
    return out;
}

CollOfScalar equelleCUDA::operator*(const CollOfScalar& lhs, const CollOfScalar& rhs) {

    CollOfScalar out = lhs;
    const double* rhs_dev = rhs.data();
    double* out_dev = out.data();

    kernelSetup s = out.setup();
    multiplication_kernel <<<s.grid, s.block>>>(out_dev, rhs_dev, out.size());
    return out;
}

CollOfScalar equelleCUDA::operator/(const CollOfScalar& lhs, const CollOfScalar& rhs) {

    CollOfScalar out = lhs;
    const double* rhs_dev = rhs.data();
    double* out_dev = out.data();

    kernelSetup s = out.setup();
    division_kernel <<<s.grid, s.block>>>(out_dev, rhs_dev, out.size());
    return out;
}

CollOfScalar equelleCUDA::operator*(const Scalar& lhs, const CollOfScalar& rhs) {
    CollOfScalar out = rhs;
    kernelSetup s = out.setup();
    multScalCollection_kernel<<<s.grid,s.block>>>(out.data(), lhs, out.size());
    return out;
}

CollOfScalar equelleCUDA::operator*(const CollOfScalar& lhs, const Scalar& rhs) {
    return (rhs * lhs);
}

CollOfScalar equelleCUDA::operator/(const CollOfScalar& lhs, const Scalar& rhs) {
    return ( (1/rhs) * lhs);
}

CollOfScalar equelleCUDA::operator/(const Scalar& lhs, const CollOfScalar& rhs) {
    CollOfScalar out = rhs;
    kernelSetup s = out.setup();
    divScalCollection_kernel<<<s.grid,s.block>>>(out.data(), lhs, out.size());
    return out;
}

CollOfScalar equelleCUDA::operator-(const CollOfScalar& arg) {
    return -1.0*arg;
}

CollOfBool equelleCUDA::operator>(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collGTcoll_kernel<<<s.grid,s.block>>>(out_ptr, lhs.data(), rhs.data(), lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator>(const CollOfScalar& lhs, const Scalar rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collGTscal_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs, lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator<(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collLTcoll_kernel<<<s.grid,s.block>>>(out_ptr, lhs.data(), rhs.data(), lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator<(const CollOfScalar& lhs, const Scalar rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collLTscal_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs, lhs.size());
    return out;
}

/////////////////////////////////////////////////////////////////////////////////
/// ----------------------- KERNEL IMPLEMENTATIONS: ---------------------------//
/////////////////////////////////////////////////////////////////////////////////



__global__ void equelleCUDA::minus_kernel(double* out, const double* rhs, const int size) {
    
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index] - rhs[index];
    }
}


__global__ void equelleCUDA::plus_kernel(double* out, const double* rhs, const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if( index < size ) {
	out[index] = out[index] + rhs[index];
    }
}

__global__ void equelleCUDA::multiplication_kernel(double* out, const double* rhs, const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index] * rhs[index];
    }
}

__global__ void equelleCUDA::division_kernel(double* out, const double* rhs, const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index] / rhs[index];
    }
}

__global__ void equelleCUDA::multScalCollection_kernel(double* out, const double scal,
						       const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index]*scal;
    }
}

__global__ void equelleCUDA::divScalCollection_kernel(double* out, const double scal,
						     const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = scal/out[index];
    }
}
						   
__global__ void equelleCUDA::comp_collGTcoll_kernel( bool* out,
						     const double* lhs,
						     const double* rhs,
						     const int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] > rhs[index];
    }
}

__global__ void equelleCUDA::comp_collGTscal_kernel( bool* out,
						     const double* lhs,
						     const double rhs,
						     const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] > rhs;
    }
}

__global__ void equelleCUDA::comp_collLTcoll_kernel( bool* out,
						     const double* lhs,
						     const double* rhs,
						     const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] < rhs[index];
    }
}

__global__ void equelleCUDA::comp_collLTscal_kernel( bool* out,
						     const double* lhs,
						     const double rhs,
						     const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] < rhs;
    }
}




// Transforming CollOfBool
std::vector<bool> equelleCUDA::cob_to_std( const CollOfBool& cob) {
    thrust::host_vector<bool> host = cob;
    return std::vector<bool>(host.begin(), host.end());
}