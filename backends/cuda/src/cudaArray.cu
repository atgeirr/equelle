#include "hip/hip_runtime.h"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/fill.h>
//#include <thrust/copy.h>
//#include <thrust/sequence.h>

#include <string>
//#include <fstream>
//#include <iterator>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <vector>
#include <iostream>

// For error exception macro:
#include <opm/core/utility/ErrorMacros.hpp>

#include "CudaArray.hpp"
#include "DeviceGrid.hpp"
#include "CollOfIndices.hpp"


// Implementation of the class CudaArray

using namespace equelleCUDA;
using namespace wrapCudaArray;

CudaArray::CudaArray() 
    : size_(0), 
      dev_values_(0),
      block_x_(0),
      grid_x_(0),
      setup_(0)
#ifdef EQUELLE_DEBUG
    , debug_vec_(0)
#endif // EQUELLE_DEBUG
{
    // Intentionally left blank
}


// Allocating memory without initialization
CudaArray::CudaArray(const int size) 
    : size_(size),
      dev_values_(0),
      block_x_(equelleCUDA::MAX_THREADS),
      grid_x_((size_ + block_x_ - 1) / block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(size,0)
#endif // EQUELLE_DEBUG
{
    cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
    checkError_("hipMalloc in CudaArray::CudaArray(int)");
#ifdef EQUELLE_DEBUG
    std::cout << "Debug mode is on!\n";
#endif // EQUELLE_DEBUG
}

CudaArray::CudaArray(const int size, const double value) 
    : size_(size),
      dev_values_(0),
      block_x_(equelleCUDA::MAX_THREADS),
      grid_x_((size_ + block_x_ - 1) / block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(size, value)
#endif // EQUELLE_DEBUG
{
    // Can not use hipMemset as it sets float values on a given
    // number of bytes.

    std::vector<double> host_vec(size_, value);

    cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
    checkError_("hipMalloc in CudaArray::CudaArray(int, double)");
        
    cudaStatus_ = hipMemcpy(dev_values_, &host_vec[0], size_*sizeof(double),
				    hipMemcpyHostToDevice);
    checkError_("hipMemcpy in CudaArray::CudaArray(int, double)");

} 


// Constructor from vector, in order to do testing
CudaArray::CudaArray(const std::vector<double>& host_vec)
    : size_(host_vec.size()),
      dev_values_(0),
      block_x_(equelleCUDA::MAX_THREADS),
      grid_x_((size_ + block_x_ - 1) / block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(host_vec)
#endif // EQUELLE_DEBUG
{
    cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
    checkError_("hipMalloc in CudaArray::CudaArray(std::vector<double>)");
    
    cudaStatus_ = hipMemcpy(dev_values_, &host_vec[0], size_*sizeof(double),
			    hipMemcpyHostToDevice);
    checkError_("hipMemcpy in CudaArray::CudaArray(std::vector<double>)");
}


// Copy constructor
CudaArray::CudaArray(const CudaArray& coll) 
    : size_(coll.size_), 
      dev_values_(0),
      grid_x_(coll.grid_x_),
      block_x_(coll.block_x_),
      setup_(size_)
#ifdef EQUELLE_DEBUG
    , debug_vec_(coll.size_, 0)
#endif // EQUELLE_DEBUG
{
    std::cout << __PRETTY_FUNCTION__ << std::endl;

    if (coll.dev_values_ != 0) {
	cudaStatus_ = hipMalloc( (void**)&dev_values_, size_*sizeof(double));
	checkError_("hipMalloc in CudaArray::CudaArray(const CudaArray&)"); 

	cudaStatus_ = hipMemcpy(dev_values_, coll.dev_values_, size_*sizeof(double),
				 hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy in CudaArray::CudaArray(const CudaArray&)");
    }
    
#ifdef EQUELLE_DEBUG
    // Copy value to the std::vector debug_vec_
    std::cout << "\tDEBUG IS ON!\n";
    if (coll.dev_values_ != 0 ) {
	cudaStatus_ = hipMemcpy( &debug_vec_[0], dev_values_, size_*sizeof(double),
				  hipMemcpyDeviceToHost );
	checkError_("hipMemcpy for DEBUG in CudaArray::CudaArray(const CudaArray&)");
	last_val = debug_vec_[size_ - 1];
    }
#endif // EQUELLE_DEBUG
}


// Copy assignment operator
CudaArray& CudaArray::operator= (const CudaArray& other) {
    std::cout << __PRETTY_FUNCTION__ << std::endl;

    // Protect agains " var = var " , self assignment
    if ( this != &other ) {

	// First idea: Make this->dev_values_ point to other.dev_values_
	// and set other.dev_values_ = 0.
	// Why is this a bad idea? We should still be able to use other.
	// THEREFORE: Need to copy the content of other.dev_values_ to
	// this->dev_values_.

	// this->dev_values will be overwritten, and can safely be freed,
	// But if the collections are of the same size (likely) we
	// Will just overwrite the old values.

	if ( this->size_ != other.size_) {

	    // If different size: Is this even allowed?
	    // Free memory:
	    cudaStatus_ = hipFree(this->dev_values_);
	    checkError_("hipFree(this->dev_values_) in CudaArray::operator=(const CudaArray&)");
	    // Allocate new memory:
	    cudaStatus_ = hipMalloc((void**)&this->dev_values_,
				     sizeof(double) * other.size_);
	    checkError_("hipMalloc(this->dev_values_) in CudaArray::operator=(const CudaArray&)");
	    
	    // Set variables depending on size_:
	    this->size_ = other.size_;
	    this->block_x_ = other.block_x_;
	    this->grid_x_ = other.grid_x_;
	}

	// Copy memory block from other to this:
	cudaStatus_ = hipMemcpy( this->dev_values_, other.dev_values_,
				  sizeof(double) * this->size_,
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(dev_values_) in CudaArray::operator=(const CudaArray&)");
	
#ifdef EQUELLE_DEBUG
	if ( debug_vec_.size() != this->size_) {
	    std::cout << "\t\tDebug vector is of size " << debug_vec_.size() << 
		" while this->size_ is " << this->size_ << "\n";
	    std::vector<double> temp(this->size_, 0);
	    cudaStatus_ = hipMemcpy( &temp[0], other.dev_values_,
				      sizeof(double) * this->size_,
				      hipMemcpyDeviceToHost);
	    checkError_("hipMemcpy(temp) in CudaArray::operator=(const CudaArray&)");
	    debug_vec_ = temp;
	}
	else {
	    cudaStatus_ = hipMemcpy( &debug_vec_[0], other.dev_values_,
				      sizeof(double) * this->size_,
				      hipMemcpyDeviceToHost);
	    checkError_("hipMemcpy(debug_vec) in CudaArray::operator=(const CudaArray&)");
	}
	last_val = debug_vec_[size_-1];
#endif // EQUELLE_DEBUG


    } // if this != &other
    
    return *this;

} // Assignment copy operator!



// Destructor:
CudaArray::~CudaArray() {
    if (dev_values_ != 0) {
	cudaStatus_ = hipFree(dev_values_);
	checkError_("hipFree in CudaArray::~CudaArray");
    }
}

#ifdef EQUELLE_DEBUG
// Debug function to get all values to host so that they can be seen by e.g. qtcreator
void CudaArray::debug() const {
    cudaStatus_ = hipMemcpy( &debug_vec_[0], dev_values_, sizeof(double)*size_,
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(debug_vec_) in CudaArray::debug()");
    last_val = debug_vec_[size_ - 1];
}
#endif // EQUELLE_DEBUG



const double* CudaArray::data() const {
    return dev_values_;
}

double* CudaArray::data() {
    return dev_values_;
}




kernelSetup CudaArray::setup() const {
    return setup_;
}

// Assumes that values are already allocated on host
std::vector<double> CudaArray::copyToHost() const
{
    // Fill host_vec with zeros:
    std::vector<double> host_vec(size_, 0);

    cudaStatus_ = hipMemcpy( &host_vec[0], dev_values_, size_*sizeof(double),
			     hipMemcpyDeviceToHost);
    checkError_("hipMemcpy in CudaArray::copyToHost");
    
    return host_vec;
}


int CudaArray::size() const
{
    return size_;
}



void CudaArray::checkError_(const std::string& msg) const {
    if ( cudaStatus_ != hipSuccess ) {
	OPM_THROW(std::runtime_error, "\nCuda error\n\t" << msg << " - Error code: " << hipGetErrorString(cudaStatus_));
    }
}






/////////////////////////////////////////////////////////////////////////////////
/// ----------------------- OPERATOR OVERLOADING: -----------------------------//
/////////////////////////////////////////////////////////////////////////////////



CudaArray equelleCUDA::operator-(const CudaArray& lhs, const CudaArray& rhs) {

    CudaArray out = lhs;
    kernelSetup s = out.setup();
    minus_kernel <<<s.grid, s.block>>>(out.data(), rhs.data(), out.size());
    return out;
}

CudaArray equelleCUDA::operator+(const CudaArray& lhs, const CudaArray& rhs) {

    CudaArray out = lhs;
    kernelSetup s = out.setup();
    plus_kernel <<<s.grid, s.block>>>(out.data(), rhs.data(), out.size());
    return out;
}

CudaArray equelleCUDA::operator*(const CudaArray& lhs, const CudaArray& rhs) {

    CudaArray out = lhs;
    kernelSetup s = out.setup();
    multiplication_kernel <<<s.grid, s.block>>>(out.data(), rhs.data(), out.size());
    return out;
}

CudaArray equelleCUDA::operator/(const CudaArray& lhs, const CudaArray& rhs) {

    CudaArray out = lhs;
    kernelSetup s = out.setup();
    division_kernel <<<s.grid, s.block>>>(out.data(), rhs.data(), out.size());
    return out;
}

CudaArray equelleCUDA::operator*(const Scalar lhs, const CudaArray& rhs) {
    CudaArray out = rhs;
    kernelSetup s = out.setup();
    scalMultColl_kernel<<<s.grid,s.block>>>(out.data(), lhs, out.size());
    return out;
}

CudaArray equelleCUDA::operator*(const CudaArray& lhs, const Scalar rhs) {
    return (rhs * lhs);
}

CudaArray equelleCUDA::operator/(const CudaArray& lhs, const Scalar rhs) {
    return ( (1/rhs) * lhs);
}

CudaArray equelleCUDA::operator/(const Scalar lhs, const CudaArray& rhs) {
    CudaArray out = rhs;
    kernelSetup s = out.setup();
    scalDivColl_kernel<<<s.grid,s.block>>>(out.data(), lhs, out.size());
    return out;
}

CudaArray equelleCUDA::operator-(const CudaArray& arg) {
    return -1.0*arg;
}


//  >
CollOfBool equelleCUDA::operator>(const CudaArray& lhs, const CudaArray& rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collGTcoll_kernel<<<s.grid,s.block>>>(out_ptr, lhs.data(), rhs.data(), lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator>(const CudaArray& lhs, const Scalar rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collGTscal_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs, lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator>(const Scalar lhs, const CudaArray& rhs) {
    CollOfBool out(rhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = rhs.setup();
    comp_scalGTcoll_kernel<<<s.grid, s.block>>>(out_ptr, lhs, rhs.data(), rhs.size());
    return out;
}


// <
CollOfBool equelleCUDA::operator<(const CudaArray& lhs, const CudaArray& rhs) {
    // if   a < b   then b > a
    return rhs > lhs;
}

CollOfBool equelleCUDA::operator<(const CudaArray& lhs, const Scalar rhs) {
    // if  a < b  then   b > a
    return rhs > lhs;
}

CollOfBool equelleCUDA::operator<(const Scalar lhs, const CudaArray& rhs) {
    // if   a < b   then b > a
    return rhs > lhs;
}


// >=
CollOfBool equelleCUDA::operator>=(const CudaArray& lhs, const CudaArray& rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collGEcoll_kernel<<<s.grid,s.block>>>(out_ptr, lhs.data(), rhs.data(), lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator>=(const CudaArray& lhs, const Scalar rhs) {
    CollOfBool out(lhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collGEscal_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs, lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator>=(const Scalar lhs, const CudaArray& rhs) {
    CollOfBool out(rhs.size());
    bool* out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = rhs.setup();
    comp_scalGEcoll_kernel<<<s.grid, s.block>>>(out_ptr, lhs, rhs.data(), rhs.size());
    return out;
}


// <= 
CollOfBool equelleCUDA::operator<=(const CudaArray& lhs, const CudaArray& rhs) {
    // if   a <= b   then b >= a
    return rhs >= lhs;
}

CollOfBool equelleCUDA::operator<=(const CudaArray& lhs, const Scalar rhs) {
    // if  a <= b  then   b >= a
    return rhs >= lhs;
}

CollOfBool equelleCUDA::operator<=(const Scalar lhs, const CudaArray& rhs) {
    // if   a <= b   then b >= a
    return rhs >= lhs;
}


// ==
CollOfBool equelleCUDA::operator==(const CudaArray& lhs, const CudaArray& rhs) {
    CollOfBool out(lhs.size());
    bool *out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collEQcoll_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs.data(), lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator==(const CudaArray& lhs, const Scalar rhs) {
    CollOfBool out(lhs.size());
    bool *out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collEQscal_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs, lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator==(const Scalar lhs, const CudaArray& rhs) {
    return (rhs == lhs);
}


// !=
CollOfBool equelleCUDA::operator!=(const CudaArray& lhs, const CudaArray& rhs) {
    CollOfBool out(lhs.size());
    bool *out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collNEcoll_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs.data(), lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator!=(const CudaArray& lhs, const Scalar rhs) {
    CollOfBool out(lhs.size());
    bool *out_ptr = thrust::raw_pointer_cast( &out[0] );
    kernelSetup s = lhs.setup();
    comp_collNEscal_kernel<<<s.grid, s.block>>>(out_ptr, lhs.data(), rhs, lhs.size());
    return out;
}

CollOfBool equelleCUDA::operator!=(const Scalar lhs, const CudaArray& rhs) {
    return (rhs != lhs);
}

/////////////////////////////////////////////////////////////////////////////////
/// ----------------------- KERNEL IMPLEMENTATIONS: ---------------------------//
/////////////////////////////////////////////////////////////////////////////////



__global__ void wrapCudaArray::minus_kernel(double* out, const double* rhs, const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index] - rhs[index];
    }
}


__global__ void wrapCudaArray::plus_kernel(double* out, const double* rhs, const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if( index < size ) {
	out[index] = out[index] + rhs[index];
    }
}

__global__ void wrapCudaArray::multiplication_kernel(double* out, const double* rhs, const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index] * rhs[index];
    }
}

__global__ void wrapCudaArray::division_kernel(double* out, const double* rhs, const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index] / rhs[index];
    }
}

__global__ void wrapCudaArray::scalMultColl_kernel(double* out, const double scal,
						       const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = out[index]*scal;
    }
}

__global__ void wrapCudaArray::scalDivColl_kernel(double* out, const double scal,
						     const int size) {
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if ( index < size ) {
	out[index] = scal/out[index];
    }
}
						   
__global__ void wrapCudaArray::comp_collGTcoll_kernel( bool* out,
							  const double* lhs,
							  const double* rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] > rhs[index];
    }
}

__global__ void wrapCudaArray::comp_collGTscal_kernel( bool* out,
							  const double* lhs,
							  const double rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] > rhs;
    }
}

__global__ void wrapCudaArray::comp_scalGTcoll_kernel( bool* out,
							  const double lhs,
							  const double* rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs > rhs[index];
    }
}

__global__ void wrapCudaArray::comp_collGEcoll_kernel( bool* out,
							  const double* lhs,
							  const double* rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] >= rhs[index];
    }
}

__global__ void wrapCudaArray::comp_collGEscal_kernel( bool* out,
							  const double* lhs,
							  const double rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs[index] >= rhs;
    }
}

__global__ void wrapCudaArray::comp_scalGEcoll_kernel( bool* out,
							  const double lhs,
							  const double* rhs,
							  const int size) 
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = lhs >= rhs[index];
    }
}


__global__ void wrapCudaArray::comp_collEQcoll_kernel( bool* out,
							  const double* lhs,
							  const double* rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = ( lhs[index] == rhs[index] );
    }
}
							
__global__ void wrapCudaArray::comp_collEQscal_kernel( bool* out,
							  const double* lhs,
							  const double rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = ( lhs[index] == rhs );
    }
}
							

__global__ void wrapCudaArray::comp_collNEcoll_kernel( bool* out,
							  const double* lhs,
							  const double* rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = ( lhs[index] != rhs[index] );
    }
}
							
__global__ void wrapCudaArray::comp_collNEscal_kernel( bool* out,
							  const double* lhs,
							  const double rhs,
							  const int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if ( index < size ) {
	out[index] = ( lhs[index] != rhs );
    }
}




// Transforming CollOfBool
std::vector<bool> equelleCUDA::cob_to_std( const CollOfBool& cob) {
    thrust::host_vector<bool> host = cob;
    return std::vector<bool>(host.begin(), host.end());
}