#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <opm/core/utility/ErrorMacros.hpp>

#include <vector>
#include <iostream>
#include <string>

#include <Eigen/Sparse>

#include "CudaMatrix.hpp"
#include "CudaArray.hpp" // kernels for scalar multiplications
#include "equelleTypedefs.hpp"

using namespace equelleCUDA;
using namespace wrapCudaMatrix;

using std::vector;

// Implementation of member functions of CudaMatrix

// Default constructor:
CudaMatrix::CudaMatrix() 
    : rows_(0),
      cols_(0),
      nnz_(0),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0)
{
    createGeneralDescription_("CudaMatrix::CudaMatrix()");
}


// Constructor from host values:
CudaMatrix::CudaMatrix( const double* val, const int* rowPtr, const int* colInd,
			const int nnz, const int rows, const int cols)
    : rows_(rows),
      cols_(cols),
      nnz_(nnz),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0)
{
    // Allocate memory and copy data to host
    cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
    checkError_("hipMalloc(csrVal_) in CudaMatrix host constructor");
    cudaStatus_ = hipMemcpy( csrVal_, val, nnz_*sizeof(double), 
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix host constructor");

    cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_ + 1)*sizeof(int));
    checkError_("hipMalloc(csrRowPtr_) in CudaMatrix host constructor");
    cudaStatus_ = hipMemcpy( csrRowPtr_, rowPtr, (rows_ + 1)*sizeof(int),
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix host constructor");

    cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
    checkError_("hipMalloc(csrColInd_) in CudaMatrix host constructor");
    cudaStatus_ = hipMemcpy( csrColInd_, colInd, nnz_*sizeof(int),
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix host constructor");

    createGeneralDescription_("CudaMatrix host constructor");
}


// Constructor from Eigen Matrix
CudaMatrix::CudaMatrix(const Eigen_M& eigen)
    : rows_(eigen.rows()),
      cols_(eigen.cols()),
      nnz_(eigen.nonZeros()),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0)
{
    // Should have a check here to ensure that the matrix is stored 
    // in a row-major format.
    
    // Opm::HelperOps creates helper matrices in column major format.
    // Copy the input to a row major matrix instead:
    Eigen::SparseMatrix<Scalar, Eigen::RowMajor> rowmajor(eigen);
    std::cout << "Rowmajor:\n" << rowmajor << "\n";

    // Allocate memory:
    cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
    checkError_("hipMalloc(csrVal_) in CudaMatrix Eigen constructor");
    cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_+1)*sizeof(int));
    checkError_("hipMalloc(csrRowPtr_) in CudaMatrix Eigen constructor");
    cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
    checkError_("hipMalloc(csrColInd_) in CudaMatrix Eigen constructor");

    // Copy arrays:
    cudaStatus_ = hipMemcpy( csrVal_, rowmajor.valuePtr(), nnz_*sizeof(double),
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix Eigen constructor");
    cudaStatus_ = hipMemcpy( csrRowPtr_, rowmajor.outerIndexPtr(), (rows_+1)*sizeof(int),
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix Eigen constructor");
    cudaStatus_ = hipMemcpy( csrColInd_, rowmajor.innerIndexPtr(), nnz_*sizeof(int),
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix Eigen constructor");

    createGeneralDescription_("CudaMatrix Eigen constructor");
} // constructor from Eigen


// Identity matrix constructor
CudaMatrix::CudaMatrix(const int size) 
    : rows_(size),
      cols_(size),
      nnz_(size),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0)
{
    // Allocate memory:
    cudaStatus_ = hipMalloc( (void**)&csrVal_, size*sizeof(double));
    checkError_("hipMalloc(csrVal_) in CudaMatrix identity matrix constructor");
    cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (size+1)*sizeof(int));
    checkError_("hipMalloc(csrRowPtr_) in CudaMatrix identity matrix constructor");
    cudaStatus_ = hipMalloc( (void**)&csrColInd_, size*sizeof(int));
    checkError_("hipMalloc(csrColInd_) in CudaMatrix identity matrix constructor");

    // Call a kernel that writes the correct data:
    kernelSetup s(size+1);
    initIdentityMatrix<<<s.grid, s.block>>>(csrVal_, csrRowPtr_, csrColInd_, nnz_);

    createGeneralDescription_("CudaMatrix identity matrix constructor");
}


// Copy constructor:
CudaMatrix::CudaMatrix(const CudaMatrix& mat)
    : rows_(mat.rows_),
      cols_(mat.cols_),
      nnz_(mat.nnz_),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0)
{
    // Copy arrays if they exist:
    if ( mat.csrVal_ != 0 ) {
	cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
	checkError_("hipMalloc(csrVal_) in CudaMatrix copy constructor");
	cudaStatus_ = hipMemcpy( csrVal_, mat.csrVal_, nnz_*sizeof(double),
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(csrVal_) in CudaMatrix copy constructor");
    }
    if ( mat.csrRowPtr_ != 0 ) {
	cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_+1)*sizeof(int));
	checkError_("hipMalloc(csrRowPtr_) in CudaMatrix copy constructor");
	cudaStatus_ = hipMemcpy( csrRowPtr_, mat.csrRowPtr_, (rows_+1)*sizeof(int),
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix copy constructor");
    }
    if ( mat.csrColInd_ != 0 ) {
	cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
	checkError_("hipMalloc(csrColInd_) in CudaMalloc copy constructor");
	cudaStatus_ = hipMemcpy( csrColInd_, mat.csrColInd_, nnz_*sizeof(int),
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(csrColInd_) in CudaMatrix copy constructor");
    }
    
    createGeneralDescription_("CudaMatrix copy constructor");
}


// Copy assignment operator:
CudaMatrix& CudaMatrix::operator= (const CudaMatrix& other) {

    // Protect against self assignment
    if ( this != &other ) {
	
	// Check if we have to reallocate memory depending on nnz:
	if ( nnz_ != other.nnz_ ) { 
	    nnz_ = other.nnz_;
	    
	    // Free and reallocate csrVal_, but only free if csrVal_ != 0
	    if ( csrVal_ != 0 ) {
		cudaStatus_ = hipFree(csrVal_);
		checkError_("hipFree(csrVal_) in CudaMatrix copy assignment operator");
	    }
	    cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
	    checkError_("hipMalloc(csrVal_) in CudaMatrix copy assignment operator");
	    
	    // Free (if nonzero) and allocate csrColInd_
	    if ( csrColInd_ != 0 ) {
		cudaStatus_ = hipFree(csrColInd_);
		checkError_("hipFree(csrColInd_) in CudaMatrix copy assignment operator");
	    }
	    cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
	    checkError_("hipMalloc(csrColInd_) in CudaMatrix copy assignment operator");
	} // if (nnz != other.nnz_)

	// Check if we have to reallocate memory depending on rows:
	if ( rows_ != other.rows_ ) {
	    rows_ = other.rows_;
	    if ( csrRowPtr_ != 0 ) {
		cudaStatus_ = hipFree(csrRowPtr_);
		checkError_("hipFree(csrRowPtr_) in CudaMatrix copy assignment operator");
	    }
	    cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_+1)*sizeof(int));
	    checkError_("hipMalloc(csrRowPtr_) in CudaMatrix copy assignment operator");
	} // if ( rows_ != other.rows_ )

	cols_ = other.cols_;
	
	// All arrays correct sizes. Copy data:
	cudaStatus_ = hipMemcpy( csrVal_, other.csrVal_, nnz_*sizeof(double),
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(csrVal_) in CudaMatrix copy assignment operator");
	cudaStatus_ = hipMemcpy( csrRowPtr_, other.csrRowPtr_, (rows_+1)*sizeof(int),
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix copy assignment operator");
	cudaStatus_ = hipMemcpy( csrColInd_, other.csrColInd_, nnz_*sizeof(int),
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(csrColInd_) in CudaMatrix copy assignment operator");

	
	// Do not have to care about description, as it is the same for all matrices!
	
    } // if ( this != &other)
    
    return *this;
}



// Destructor
CudaMatrix::~CudaMatrix() {
    // Free pointers if not zero.
    if ( csrVal_ != 0 ) {
	cudaStatus_ = hipFree(csrVal_);
	checkError_("hipFree(csrVal_) in CudaMatrix::~CudaMatrix");
    }
    if ( csrRowPtr_ != 0 ) {
	cudaStatus_ = hipFree(csrRowPtr_);
	checkError_("hipFree(csrRowPtr_) in CudaMatrix::~CudaMatrix");
    }
    if ( csrColInd_ != 0 ) {
	cudaStatus_ = hipFree(csrColInd_);
	checkError_("hipFree(csrColInd_) in CudaMatrix::~CudaMatrix");
    }

    // Destroy description_ 
    sparseStatus_ = hipsparseDestroyMatDescr( description_ );
    checkError_("hipsparseDestroyMatDescr() in CudaMatrix::~CudaMatrix()");

    std::cout << "Freeing matrix\n";

}





int CudaMatrix::nnz() const {
    return nnz_;
}
int CudaMatrix::rows() const {
    return rows_;
}
int CudaMatrix::cols() const {
    return cols_;
}


// Copy to host:
hostMat CudaMatrix::toHost() const {
    if ( (csrVal_ == 0) || (csrRowPtr_ == 0) || (csrColInd_ == 0) ) {
	OPM_THROW(std::runtime_error, "Tried to copy matrix to host, but the pointers are (" << csrVal_ << "," << csrRowPtr_ << "," << csrColInd_ << ")");
    }

    vector<double> v(nnz_, -1);
    cudaStatus_ = hipMemcpy( &v[0], csrVal_, nnz_*sizeof(double),
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix::toHost()");
    
    vector<int> rp(rows_+1, -1);
    cudaStatus_ = hipMemcpy( &rp[0], csrRowPtr_, (rows_+1)*sizeof(int),
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix::toHost()");
    
    vector<int> ci(nnz_, -1);
    cudaStatus_ = hipMemcpy( &ci[0], csrColInd_, nnz_*sizeof(int),
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix::toHost()");
    
    hostMat out;
    out.vals = v;
    out.rowPtr = rp;
    out.colInd = ci;
    out.nnz = nnz_;
    out.rows = rows_;
    out.cols = cols_;
    return out;
}


// Error checking:
void CudaMatrix::checkError_(const std::string& msg) const {
    if ( cudaStatus_ != hipSuccess) {
	OPM_THROW(std::runtime_error, "\nCuda error\n\t" << msg << " - Error code: "<< hipGetErrorString(cudaStatus_) );
    }
    if ( sparseStatus_ != HIPSPARSE_STATUS_SUCCESS ) {
	OPM_THROW(std::runtime_error, "\ncusparse error\n\t" << msg << " - Error code: " << sparseStatus_);
    }
}

void CudaMatrix::createGeneralDescription_(const std::string& msg) {
    sparseStatus_ = hipsparseCreateMatDescr( &description_ );
    checkError_("hipsparseCreateMatDescr() in " + msg);
    
    sparseStatus_ = hipsparseSetMatType(description_, HIPSPARSE_MATRIX_TYPE_GENERAL);
    checkError_("hipsparseSetMatType() in " + msg);
    sparseStatus_ = hipsparseSetMatIndexBase(description_, HIPSPARSE_INDEX_BASE_ZERO);
    checkError_("hipsparseSetMatIndexBase() in " + msg);
}



// Operator +
CudaMatrix equelleCUDA::operator+(const CudaMatrix& lhs, const CudaMatrix& rhs) {
    return cudaMatrixSum(lhs, rhs, 1.0);
}

CudaMatrix equelleCUDA::operator-(const CudaMatrix& lhs, const CudaMatrix& rhs) {
    return cudaMatrixSum(lhs, rhs, -1.0);
}


CudaMatrix equelleCUDA::cudaMatrixSum(const CudaMatrix& lhs, 
				      const CudaMatrix& rhs,
				      const double beta) {
  
    if ( (lhs.rows_ != rhs.rows_) || (lhs.cols_ != rhs.cols_) ) {
    	OPM_THROW(std::runtime_error, "Error in CudaMatrix + CudaMatrix\n" << "\tMatrices of different size.\n" << "\tlhs: " << lhs.rows_ << " x " << lhs.cols_ << "\n" << "\trhs: " << rhs.rows_ << " x " << rhs.cols_ << ".");
    }

    // Create an empty matrix. Need to set rows, cols, nnz, and allocate arrays!
    CudaMatrix out;
    out.rows_ = lhs.rows_;
    out.cols_ = lhs.cols_;

    // Addition in two steps
    //    1) Find nonzero pattern of output
    //    2) Add matrices.

    // 1) Find nonzero pattern:
    // Allocate rowPtr:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrRowPtr_, (out.rows_+1)*sizeof(int));
    out.checkError_("hipMalloc(out.csrRowPtr_) in cudaMatrixSum()");

    // The following code for finding number of non-zeros is
    // taken from the Nvidia cusparse documentation, section 9.1
    // Only additions are the error checking.
    // The documentation has a typo, as it says cusparseSetPointerNode, not Mode.
    int *nnzTotalDevHostPtr = &out.nnz_;
    out.sparseStatus_ = hipsparseSetPointerMode( CUSPARSE, HIPSPARSE_POINTER_MODE_HOST);
    out.checkError_("hipsparseSetPointerMode() in cudaMatrixSum()");
    out.sparseStatus_ = hipsparseXcsrgeamNnz( CUSPARSE, out.rows_, out.cols_,
					     lhs.description_, lhs.nnz_,
					     lhs.csrRowPtr_, lhs.csrColInd_,
					     rhs.description_, rhs.nnz_,
					     rhs.csrRowPtr_, rhs.csrColInd_,
					     out.description_, out.csrRowPtr_,
					     nnzTotalDevHostPtr);
    out.checkError_("hipsparseXcsrgeamNnz() in cudaMatrixSum()");
    if ( nnzTotalDevHostPtr != NULL) {
	out.nnz_ = *nnzTotalDevHostPtr;
    } else {
	out.cudaStatus_ = hipMemcpy( &out.nnz_, out.csrRowPtr_ + out.rows_,
				      sizeof(int), hipMemcpyDeviceToHost);
	out.checkError_("hipMemcpy(out.csrRowPtr_ + rows_) in cudaMatrixSum()");
	int baseC;
	out.cudaStatus_ = hipMemcpy( &baseC, out.csrRowPtr_, sizeof(int),
				      hipMemcpyDeviceToHost);
	out.checkError_("hipMemcpy(&baseC) in cudaMatrixSum()");
	out.nnz_ -= baseC;
    }

    std::cout << "New nnz = " << out.nnz_ << std::endl;
    
    // Allocate the other two arrays:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrVal_, out.nnz_*sizeof(double));
    out.checkError_("hipMalloc(out.csrVal_) in cudaMatrixSum()");
    out.cudaStatus_ = hipMalloc( (void**)&out.csrColInd_, out.nnz_*sizeof(int));
    out.checkError_("hipMalloc(out.csrColInd_) in cudaMatrixSum()");
    
    // 2) Add matrices
    // Need to create alpha and beta:
    const double alpha = 1.0;
    //double beta = 1.0;
    out.sparseStatus_ = hipsparseDcsrgeam(CUSPARSE, out.rows_, out.cols_,
					 &alpha,
					 lhs.description_, lhs.nnz_,
					 lhs.csrVal_, lhs.csrRowPtr_, lhs.csrColInd_,
					 &beta,
					 rhs.description_, rhs.nnz_,
					 rhs.csrVal_, rhs.csrRowPtr_, rhs.csrColInd_,
					 out.description_,
					 out.csrVal_, out.csrRowPtr_, out.csrColInd_);
    out.checkError_("cusparseDcsrgream() in cudaMatrixSum()");

    return out;

} // cudaMatrixSum



CudaMatrix equelleCUDA::operator*(const CudaMatrix& lhs, const CudaMatrix& rhs) {

    if ( lhs.cols_ != rhs.rows_ ) {
	OPM_THROW(std::runtime_error, "Error in CudaMatrix * CudaMatrix\n" << "\tMatrices of illegal sizes.\n" << "\tlhs.cols_ = " << lhs.cols_ << "\n\trhs.rows_ = " << rhs.rows_);
    }

    // Create an empty matrix. Need to set rows, cols, nnz, and allocate arrays!
    CudaMatrix out;
    out.rows_ = lhs.rows_;
    out.cols_ = rhs.cols_;

    // Addition in two steps
    //    1) Find nonzero pattern of output
    //    2) Multiply matrices.

    // 1) Find nonzero pattern of output
    // Allocate rowPtr:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrRowPtr_, (out.rows_+1)*sizeof(int));
    out.checkError_("hipMalloc(out.csrRowPtr_) in CudaMatrix operator +");

    // The following code for finding number of non-zeros is
    // taken from the Nvidia cusparse documentation, section 9.2
    // Only additions are the error checking.
    int *nnzTotalDevHostPtr = &out.nnz_;
    out.sparseStatus_ = hipsparseSetPointerMode(CUSPARSE, HIPSPARSE_POINTER_MODE_HOST);
    out.checkError_("hipsparseSetPointerMode() in CudaMatrix operator *");
    out.sparseStatus_ = hipsparseXcsrgemmNnz( CUSPARSE, 
					     HIPSPARSE_OPERATION_NON_TRANSPOSE,
					     HIPSPARSE_OPERATION_NON_TRANSPOSE,
					     out.rows_, out.cols_, lhs.cols_,
					     lhs.description_, lhs.nnz_,
					     lhs.csrRowPtr_, lhs.csrColInd_,
					     rhs.description_, rhs.nnz_,
					     rhs.csrRowPtr_, rhs.csrColInd_,
					     out.description_,
					     out.csrRowPtr_, nnzTotalDevHostPtr);
    out.checkError_("hipsparseXcsrgemmNnz() in CudaMatrix operator *");
    if ( nnzTotalDevHostPtr != NULL ) {
	out.nnz_ = *nnzTotalDevHostPtr;
    } else {
	int baseC;
	out.cudaStatus_ = hipMemcpy(&out.nnz_, out.csrRowPtr_ + out.rows_,
				     sizeof(int), hipMemcpyDeviceToHost);
	out.checkError_("hipMemcpy(out.csrRowPtr_ + out.rows_) in CudaMatrix operator *");
	out.cudaStatus_ = hipMemcpy(&baseC, out.csrRowPtr_, sizeof(int),
				     hipMemcpyDeviceToHost);
	out.checkError_("hipMemcpy(baseC) in CudaMatrix operator *");
	out.nnz_ -= baseC;
    }

    std::cout << "New nnz: " << out.nnz_ << "\n";
    
    // Allocate the other two arrays:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrVal_, out.nnz_*sizeof(double));
    out.checkError_("hipMalloc(out.csrVal_) in CudaMatrix operator *");
    out.cudaStatus_ = hipMalloc( (void**)&out.csrColInd_, out.nnz_*sizeof(int));
    out.checkError_("hipMalloc(out.csrColInd_) in CudaMatrix operator *");
    
    // 2) Multiply the matrices:
    out.sparseStatus_ = hipsparseDcsrgemm(CUSPARSE,
					 HIPSPARSE_OPERATION_NON_TRANSPOSE,
					 HIPSPARSE_OPERATION_NON_TRANSPOSE,
					 out.rows_, out.cols_, lhs.cols_,
					 lhs.description_, lhs.nnz_,
					 lhs.csrVal_, lhs.csrRowPtr_, lhs.csrColInd_,
					 rhs.description_, rhs.nnz_,
					 rhs.csrVal_, rhs.csrRowPtr_, rhs.csrColInd_,
					 out.description_,
					 out.csrVal_, out.csrRowPtr_, out.csrColInd_);
    out.checkError_("hipsparseDcsrgemm() in CudaMatrix operator *");
    
    return out;
} // operator *

// Scalar multiplications with matrix:
CudaMatrix equelleCUDA::operator*(const CudaMatrix& lhs, const Scalar rhs) {
    CudaMatrix out(lhs);
    kernelSetup s(out.nnz_);
    wrapCudaArray::scalMultColl_kernel<<<s.grid, s.block>>>(out.csrVal_,
							    rhs,
							    out.nnz_);
    return out;
}

CudaMatrix equelleCUDA::operator*(const Scalar lhs, const CudaMatrix& rhs) {
    CudaMatrix out(rhs);
    kernelSetup s(out.nnz_);
    wrapCudaArray::scalMultColl_kernel<<<s.grid, s.block>>>(out.csrVal_,
							    lhs,
							    out.nnz_);
    return out;
}


__global__ void wrapCudaMatrix::initIdentityMatrix(double* csrVal,
						   int* csrRowPtr,
						   int* csrColInd,
						   const int nnz)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if ( i < nnz + 1) {
	csrRowPtr[i] = i;
	if (i < nnz) {
	    csrVal[i] = 1.0;
	    csrColInd[i] = i;
	}
    }
}