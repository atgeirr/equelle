#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <opm/core/utility/ErrorMacros.hpp>

#include <vector>
#include <iostream>
#include <string>

#include "CudaMatrix.hpp"


using namespace equelleCUDA;
using std::vector;

// Implementation of member functions of CudaMatrix

// Default constructor:
CudaMatrix::CudaMatrix() 
    : rows_(0),
      cols_(0),
      nnz_(0),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess)
{
    // Intentionally left blank
}


// Constructor from host values:
CudaMatrix::CudaMatrix( const double* val, const int* rowPtr, const int* colInd,
			const int nnz, const int rows, const int cols)
    : rows_(rows),
      cols_(cols),
      nnz_(nnz),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess)
{
    // Allocate memory and copy data to host
    cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
    checkError_("hipMalloc(csrVal_) in CudaMatrix host constructor");
    cudaStatus_ = hipMemcpy( csrVal_, val, nnz_*sizeof(double), 
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix host constructor");

    cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_ + 1)*sizeof(int));
    checkError_("hipMalloc(csrRowPtr_) in CudaMatrix host constructor");
    cudaStatus_ = hipMemcpy( csrRowPtr_, rowPtr, (rows_ + 1)*sizeof(int),
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix host constructor");

    cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
    checkError_("hipMalloc(csrColInd_) in CudaMatrix host constructor");
    cudaStatus_ = hipMemcpy( csrColInd_, colInd, nnz_*sizeof(int),
			      hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix host constructor");
}


// Destructor
CudaMatrix::~CudaMatrix() {
    // Free pointers if not zero.
    if ( csrVal_ != 0 ) {
	cudaStatus_ = hipFree(csrVal_);
	checkError_("hipFree(csrVal_) in CudaMatrix::~CudaMatrix");
    }
    if ( csrRowPtr_ != 0 ) {
	cudaStatus_ = hipFree(csrRowPtr_);
	checkError_("hipFree(csrRowPtr_) in CudaMatrix::~CudaMatrix");
    }
    if ( csrColInd_ != 0 ) {
	cudaStatus_ = hipFree(csrColInd_);
	checkError_("hipFree(csrColInd_) in CudaMatrix::~CudaMatrix");
    }
}

int CudaMatrix::nnz() const {
    return nnz_;
}
int CudaMatrix::rows() const {
    return rows_;
}
int CudaMatrix::cols() const {
    return cols_;
}


// Copy to host:
hostMat CudaMatrix::toHost() const {
    if ( (csrVal_ == 0) || (csrRowPtr_ == 0) || (csrColInd_ == 0) ) {
	OPM_THROW(std::runtime_error, "Tried to copy matrix to host, but the pointers are (" << csrVal_ << "," << csrRowPtr_ << "," << csrColInd_ );
    }

    vector<double> v(nnz_, -1);
    cudaStatus_ = hipMemcpy( &v[0], csrVal_, nnz_*sizeof(double),
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix::toHost()");
    
    vector<int> rp(rows_+1, -1);
    cudaStatus_ = hipMemcpy( &rp[0], csrRowPtr_, (rows_+1)*sizeof(int),
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix::toHost()");
    
    vector<int> ci(nnz_, -1);
    cudaStatus_ = hipMemcpy( &ci[0], csrColInd_, nnz_*sizeof(int),
			      hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix::toHost()");
    
    hostMat out;
    out.vals = v;
    out.rowPtr = rp;
    out.colInd = ci;
    out.nnz = nnz_;
    out.rows = rows_;
    out.cols = cols_;
    return out;
}


// Error checking:
void CudaMatrix::checkError_(const std::string& msg) const {
    if ( cudaStatus_ != hipSuccess) {
	OPM_THROW(std::runtime_error, "\nCuda error\n\t" << msg << " - Error code: "<< hipGetErrorString(cudaStatus_) );
    }
    if ( sparseStatus_ != HIPSPARSE_STATUS_SUCCESS ) {
	OPM_THROW(std::runtime_error, "\ncusparse error\n\t" << msg << " - Error code: " << sparseStatus_);
    }
}