#include "CusparseManager.hpp"

using namespace equelleCUDA;

CusparseManager::CusparseManager()
{
    //std::cout << "CusparseManager constructed." << std::endl;
    // Set up cuSPARSE
    hipsparseCreate(&cusparseHandle_);
    hipsparseSetPointerMode(cusparseHandle_, HIPSPARSE_POINTER_MODE_HOST);
}

CusparseManager::~CusparseManager()
{
    //std::cout << "CusparseManager destroyed." << std::endl;
    hipsparseDestroy(cusparseHandle_);
}

/// Using the Meyers singleton pattern.
CusparseManager& CusparseManager::instance()
{
    static CusparseManager s;
    return s;
}

CudaMatrix CusparseManager::matrixMultiply(const CudaMatrix& lhs, const CudaMatrix& rhs)
{
    return instance().gemm(lhs, rhs);
}

CudaMatrix CusparseManager::gemm(const CudaMatrix& lhs, const CudaMatrix& rhs)
{

    // Declare output matrix and set its dimensions (if lhs and rhs are compatible).
    CudaMatrix out;
    int innerSize = out.confirmMultSize(lhs, rhs);

    // Allocate row pointer array.
    out.cudaStatus_ = hipMalloc( (void**)&out.csrRowPtr_, (out.rows_+1)*sizeof(int));
    out.checkError_("hipMalloc(out.csrRowPtr_) in CusparseManager::gemm()");
    
    // Find the resulting non-zero pattern
    out.sparseStatus_ = hipsparseXcsrgemmNnz( cusparseHandle_, 
                         lhs.operation_, rhs.operation_,
                         out.rows_, out.cols_, innerSize,
                         lhs.description_, lhs.nnz_,
                         lhs.csrRowPtr_, lhs.csrColInd_,
                         rhs.description_, rhs.nnz_,
                         rhs.csrRowPtr_, rhs.csrColInd_,
                         out.description_,
                         out.csrRowPtr_, &out.nnz_);
    out.checkError_("hipsparseXcsrgemmNnz() in CusparseManager::gemm()");

    // Allocate value array and column index array.
    out.cudaStatus_ = hipMalloc( (void**)&out.csrVal_, out.nnz_*sizeof(double));
    out.checkError_("hipMalloc(out.csrVal_) in CusparseManager::gemm()");
    out.cudaStatus_ = hipMalloc( (void**)&out.csrColInd_, out.nnz_*sizeof(int));
    out.checkError_("hipMalloc(out.csrColInd_) in CusparseManager::gemm()");

    // Perform the multiplication.
    out.sparseStatus_ = hipsparseDcsrgemm(cusparseHandle_,
                     lhs.operation_, rhs.operation_,
                     out.rows_, out.cols_, innerSize,
                     lhs.description_, lhs.nnz_,
                     lhs.csrVal_, lhs.csrRowPtr_, lhs.csrColInd_,
                     rhs.description_, rhs.nnz_,
                     rhs.csrVal_, rhs.csrRowPtr_, rhs.csrColInd_,
                     out.description_,
                     out.csrVal_, out.csrRowPtr_, out.csrColInd_);
    out.checkError_("hipsparseDcsrgemm() in CusparseManager::gemm()");
    
    return out;
}